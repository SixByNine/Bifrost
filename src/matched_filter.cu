#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include <iostream>
#include <fstream>
#include <pipeline/matched_filter.hpp>
#include <pipeline/strided_range.hpp>
#include <utils/exceptions.hpp>

#include <thrust/device_vector.h>
#include <thrust/transform_scan.h>

// TODO: Add error checking to the methods in here
template<typename T>
class MatchedFilterPlan_impl {
	thrust::device_vector<T> m_scanned;
	hd_size                  m_max_width;
	
public:
	hd_error prep(const T* d_in, hd_size count, hd_size max_width) {
		m_max_width = max_width;
		
		thrust::device_ptr<const T> d_in_begin(d_in);
		thrust::device_ptr<const T> d_in_end(d_in + count);
		
		// Note: One extra element so that we include the final value
		m_scanned.resize(count + 1);
		thrust::exclusive_scan(d_in_begin, d_in_end + 1,
		                       m_scanned.begin());

//		std::ofstream scan_out("scan_out.dat", std::ofstream::out | std::ofstream::trunc);
//		std::ofstream orig_data("orig_data.dat", std::ofstream::out | std::ofstream::trunc);


/*		for (size_t i = 0; i < count + 1; i++)
		{
			orig_data << *(d_in_begin + i) << std::endl;
			scan_out << m_scanned[i] << std::endl;
		}
		scan_out.close(); */

		return HD_NO_ERROR;
	}

	// Note: This writes div_round_up(count + 1 - max_width, tscrunch) values to d_out
	//         with a relative starting offset of max_width/2
	// Note: This does not apply any normalisation to the output
	hd_error exec(T* d_out, hd_size filter_width, hd_size tscrunch=1) {
		// TODO: Check that prep( ) has been called
		// TODO: Check that width <= m_max_width
		
		thrust::device_ptr<T> d_out_begin(d_out);
		
		hd_size offset    = m_max_width / 2;

		// ahead and behind are the same except for the case when filter_width = 1
		hd_size ahead     = (filter_width-1)/2+1;   // Divide and round up
		hd_size behind    = filter_width / 2;       // Divide and round down
		hd_size out_count = m_scanned.size() - m_max_width;
		

		hd_size stride = tscrunch;

//		std::cout << "filter width = " << filter_width << std::endl;
//		std::cout << "ahead = " << ahead << std::endl;
//		std::cout << "behind = " << behind << std::endl;
//		std::cout << "stride = " << stride << std::endl;

		typedef typename thrust::device_vector<T>::iterator Iterator;

		// Striding through the scanned array has the same effect as tscrunching
		// TODO: Think about this carefully. Does it do exactly what we want?
		strided_range<Iterator> in_range1(m_scanned.begin()+offset + ahead,
		                                  m_scanned.begin()+offset + ahead + out_count,
		                                  stride);
		strided_range<Iterator> in_range2(m_scanned.begin()+offset - behind,
		                                  m_scanned.begin()+offset - behind + out_count,
		                                  stride);


		thrust::transform(in_range1.begin(), in_range1.end(),
		                  in_range2.begin(),
		                  d_out_begin,
		                  thrust::minus<T>());

/*		if (filter_width == 512)
		{
			std::ofstream first_range("first_range.dat", std::ofstream::out | std::ofstream::trunc);
			std::ofstream second_range("second_range.dat", std::ofstream::out | std::ofstream::trunc);
			std::ofstream difference("difference.dat", std::ofstream::out | std::ofstream::trunc);

			for (size_t i = 0; i < out_count; i++)
			{
				first_range << *(in_range1.begin() + i) << std::endl;
				second_range << *(in_range2.begin() + i) << std::endl;
				difference << *(d_out_begin +i) << std::endl;
			}

			first_range.close();
			second_range.close();
			difference.close();
			
			std::cout << "Saved some data...";
			exit(0);
		} */
		return HD_NO_ERROR;
	}
};

// Public interface (wrapper for implementation)
template<typename T>
MatchedFilterPlan<T>::MatchedFilterPlan() : m_impl(new MatchedFilterPlan_impl<T>) {}
template<typename T>
hd_error MatchedFilterPlan<T>::prep(const T* d_in, hd_size count,
                                    hd_size max_width) {
	return m_impl->prep(d_in, count, max_width);
	//return (*this)->prep(d_in, count, max_width);
}
template<typename T>
hd_error MatchedFilterPlan<T>::exec(T* d_out, hd_size filter_width,
                                    hd_size tscrunch) {
	return m_impl->exec(d_out, filter_width, tscrunch);
}

// Explicit template instantiations for types used by other compilation units
template struct MatchedFilterPlan<hd_float>;
template struct MatchedFilterPlan<int>;
